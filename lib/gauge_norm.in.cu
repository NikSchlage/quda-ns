#include "hip/hip_runtime.h"
#include <gauge_field_order.h>
#include <instantiate.h>

namespace quda {

  using namespace gauge;

  enum norm_type_ {
    NORM1,
    NORM2,
    ABS_MAX,
    ABS_MIN
  };

  template <typename reg_type, typename real, int Nc, QudaGaugeFieldOrder order>
  real_t norm(const GaugeField &u, int d, norm_type_ type) {
    real_t norm_ = 0.0;
    switch(type) {
    case   NORM1: norm_ = FieldOrder<reg_type,Nc,1,order,true,real>(const_cast<GaugeField &>(u)).norm1(d);   break;
    case   NORM2: norm_ = FieldOrder<reg_type,Nc,1,order,true,real>(const_cast<GaugeField &>(u)).norm2(d);   break;
    case ABS_MAX: norm_ = FieldOrder<reg_type,Nc,1,order,true,real>(const_cast<GaugeField &>(u)).abs_max(d); break;
    case ABS_MIN: norm_ = FieldOrder<reg_type,Nc,1,order,true,real>(const_cast<GaugeField &>(u)).abs_min(d); break;
    }
    return norm_;
  }

  template <typename T, bool fixed> struct type_mapper {
    using reg_t = typename mapper<T>::type;
    using store_t = T;
  };

  // fixed-point single-precision field
  template <> struct type_mapper<float, true> {
    using reg_t = float;
    using store_t = int;
  };

  template <typename T, bool fixed, int nColor>
  real_t norm(const GaugeField &u, int d, norm_type_ type) {
    using reg_t = typename type_mapper<T, fixed>::reg_t;
    using store_t = typename type_mapper<T, fixed>::store_t;
    real_t norm_ = 0.0;
    switch (u.FieldOrder()) {
    case QUDA_FLOAT2_GAUGE_ORDER: norm_ = norm<reg_t, store_t, nColor, QUDA_FLOAT2_GAUGE_ORDER>(u, d, type); break;
    case QUDA_QDP_GAUGE_ORDER:    norm_ = norm<reg_t, store_t, nColor, QUDA_QDP_GAUGE_ORDER>(u, d, type); break;
    case QUDA_MILC_GAUGE_ORDER:   norm_ = norm<reg_t, store_t, nColor, QUDA_MILC_GAUGE_ORDER>(u, d, type); break;
    default: errorQuda("Gauge field %d order not supported", u.Order());
    }
    return norm_;
  }

  template <int...> struct IntList { };

  template <typename T, bool fixed, int nColor, int...N>
  real_t norm(const GaugeField &u, int d, norm_type_ type, IntList<nColor, N...>)
  {
    real_t norm_ = 0.0;
    if (u.Ncolor() == 3 && u.Ncolor() == nColor) {
      norm_ = norm<T, fixed, nColor>(u, d, type);
    } else if (u.Ncolor() > 3 && u.Ncolor() / 2 == nColor) {
      norm_ = norm<T, fixed, 2 * nColor>(u, d, type); // factor of two to account for spin with MG fields
    } else {
      if constexpr (sizeof...(N) > 0) {
        norm_ = norm<T, fixed>(u, d, type, IntList<N...>());        
      } else {
        errorQuda("Nc = %d has not been instantiated", u.Ncolor());
      }
    }
    return norm_;
  }

  template <typename T> struct Norm {
    Norm(const GaugeField &u, real_t &nrm, int d, bool fixed, norm_type_ type)
    {
      if (fixed && u.Precision() > QUDA_SINGLE_PRECISION)
        errorQuda("Fixed point override only enabled for 8-bit, 16-bit and 32-bit fields");

      if (fixed) nrm = norm<T,  true>(u, d, type, IntList<@QUDA_MULTIGRID_NC_NVEC_LIST@>());
      else       nrm = norm<T, false>(u, d, type, IntList<@QUDA_MULTIGRID_NC_NVEC_LIST@>());
    }
  };

  real_t GaugeField::norm1(int d, bool fixed) const {
    if (reconstruct != QUDA_RECONSTRUCT_NO) errorQuda("Unsupported reconstruct=%d", reconstruct);
    real_t nrm = 0.0;
    instantiatePrecision<Norm>(*this, nrm, d, fixed, NORM1);
    return nrm;
  }

  real_t GaugeField::norm2(int d, bool fixed) const {
    if (reconstruct != QUDA_RECONSTRUCT_NO) errorQuda("Unsupported reconstruct=%d", reconstruct);
    real_t nrm = 0.0;
    instantiatePrecision<Norm>(*this, nrm, d, fixed, NORM2);
    return nrm;
  }

  real_t GaugeField::abs_max(int d, bool fixed) const {
    if (reconstruct != QUDA_RECONSTRUCT_NO) errorQuda("Unsupported reconstruct=%d", reconstruct);
    real_t nrm = 0.0;
    instantiatePrecision<Norm>(*this, nrm, d, fixed, ABS_MAX);
    return nrm;
  }

  real_t GaugeField::abs_min(int d, bool fixed) const {
    if (reconstruct != QUDA_RECONSTRUCT_NO) errorQuda("Unsupported reconstruct=%d", reconstruct);
    real_t nrm = std::numeric_limits<double>::infinity();
    instantiatePrecision<Norm>(*this, nrm, d, fixed, ABS_MIN);
    return nrm;
  }

} // namespace quda
